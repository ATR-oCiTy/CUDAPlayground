#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "stdio.h"

__global__ void hello_cuda() {
	printf("Hello CUDA world \n");
}

int main() {
	
	int nx, ny, nz;
	nx = 16;
	ny = 4;
	nz = 1;

	dim3 block(8, 2, 1);
	dim3 grid(nx/block.x, ny/block.y, nz/block.z);

	hello_cuda << <grid, block >> > (); //Async function call
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
