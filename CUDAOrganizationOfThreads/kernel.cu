#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "stdio.h"

__global__ void print_details() {

	printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d \nblockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d \nblockDim.x : %d, blockDim.y : %d \ngridDim.x : %d, gridDim.y : %d \n\n", 
		threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

int main() {
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);

	print_details << <grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}